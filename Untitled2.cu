#include "hip/hip_runtime.h"

#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void square (float * d_out, float * d_in)
{
  int idx=threadIdx.x;
  float f=d_in[idx];
  d_out[idx]=f*f;
}

int main(int argc, char ** argv)
{
  const int size=64;
  float h_in[64]; float * h_out[64];
  int i;
  const int bytes=64* sizeof(float);

  for(i=0;i<64;i++)
  {
    d_in[i]=float(i);
  }

  float * d_in;
  float * d_out;

  hipMalloc((void **) & d_in, bytes);
  hipMalloc((void **) & d_out, bytes);

  hipMemcpy(d_in, h_in,bytes,hipMemcpyHostToDevice);

  square<<<1, size>>>(d_out,d_in);

  hipMemcpy(h_out,d_out,bytes,hipMemcpyDeviceToHost);

  for(i=0;i<64;i++)
  {
    printf("%f ", h_out[i]);

  }

  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
